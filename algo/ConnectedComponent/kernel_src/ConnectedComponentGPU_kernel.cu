#include "hip/hip_runtime.h"
#include "ConnectedComponentGPU_kernel.h"

__global__ void MSGApply_kernel(Vertex *vSet, int numOfInitV, int *initVSet, int *vValues,
	int numOfMsg, int *mDstSet, int *mInitVIndexSet, int *mValueSet)
{
	int tid = threadIdx.x;

	if(tid < numOfMsg)
	{
		int vID = mDstSet[tid];

		if(vValues[vID] > mValueSet[tid])
		{
			vValues[vID] = mValueSet[tid];
			vSet[vID].isActive = true;
		}

		else;
	}
}

hipError_t MSGApply_kernel_exec(Vertex *vSet, int numOfInitV, int *initVSet, int *vValues,
	int numOfMsg, int *mDstSet, int *mInitVIndexSet, int *mValueSet)
{
	hipError_t err = hipSuccess;
	
	MSGApply_kernel<<<1, NUMOFGPUCORE>>>(vSet, numOfInitV, initVSet, vValues, numOfMsg, mDstSet, mInitVIndexSet, mValueSet);
    err = hipGetLastError();

	hipDeviceSynchronize();
	
	return err;
}

__global__ void MSGGenMerge_kernel(int *mValues,
	Vertex *vSet, int numOfInitV, int *initVSet, int *vValues,
	int numOfEdge, Edge *eSet)
{
	int tid = threadIdx.x;

	if(tid < numOfEdge)
	{
		int vID = -1;
		if(vSet[eSet[tid].src].isActive) vID = eSet[tid].dst;

		if(vID != -1)
		{
			for(int i = 0; i < numOfInitV; i++)
				atomicMin(&mValues[vID], vValues[eSet[tid].src] + eSet[tid].weight);
		}
		else;
	}
}

hipError_t MSGGenMerge_kernel_exec(int *mValues,
	Vertex *vSet, int numOfInitV, int *initVSet, int *vValues,
	int numOfEdge, Edge *eSet)
{
	hipError_t err = hipSuccess;

	MSGGenMerge_kernel<<<1, NUMOFGPUCORE>>>(mValues, vSet, numOfInitV, initVSet, vValues, numOfEdge, eSet);
	err = hipGetLastError();

	hipDeviceSynchronize();
	
	return err;
}